#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <limits.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gpudummy.cuh"
#include "gputools.cuh"
#include "interface.h"
#include "main.h"

int main(int argc, char **argv){

    checkargs(argc, argv, 6, "./prog <dev> <repeats> <methods> <density> <seed>\nmethods:\n0 bounding box\n1 lambda\n2 lambda-tc\n3 lambda-tc_opt\n");

    unsigned int dev        = atoi(argv[1]);
    unsigned int REPEATS    = atoi(argv[2]);
    unsigned int method     = atoi(argv[3]);
    unsigned int seed       = atof(argv[5]);
    double       density    = atof(argv[4]);

    hipSetDevice(dev);

#ifdef DEBUG
    print_gpu_specs(dev);
    printf("maxlong %lu\n", LONG_MAX);
#endif
    last_cuda_error("hipSetDevice");

    srand(seed);

    statistics stat = gpudummy(method, REPEATS, density);
#ifdef DEBUG
    printf("\x1b[1m"); fflush(stdout);
    printf("results: mean=%f[s]   var=%f   stdev=%f   sterr=%f\n", stat.mean, stat.variance, stat.stdev, stat.sterr);
    printf("\x1b[0m"); fflush(stdout);
#endif
}
