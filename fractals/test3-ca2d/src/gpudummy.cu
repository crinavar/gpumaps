#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <math.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cassert>

#include "gputools.cu"
#include "gpudummy.cuh"
#include "interface.h"
#include "kernels.cuh"

using namespace nvcuda;

unsigned int REPEATS; // Lazy programming
unsigned int WSIZE;  // Lazy programming

statistics gpudummy(unsigned int method, unsigned int repeats, double density){

    RunningStat* r;
    statistics stat;

    WSIZE               = min(BSIZE1D, WARPSIZE);
    unsigned long n     = (int)ceil(pow(2, RLEVEL));        // Linear size
    unsigned long rb    = max((int)(RLEVEL - BPOWER), 0);   // BlockTiled level
    unsigned long nb    = (int)(1 << rb);                   // N elements in BTiled level

    REPEATS = repeats;

#ifdef DEBUG
    printf("BPOWER=%i, BSIZE=%i, WSIZE=%i, scale r=%i, block-scale rb = %i   nb = %i\n", BPOWER, BSIZE2D, WSIZE, RLEVEL, rb, nb);

    printf("[Lambda (inverse)]\n");
#endif

    MTYPE *hdata, *ddata;
    MTYPE *hmat, *dmat1, *dmat2;
	unsigned long msize, trisize;

#ifdef DEBUG
    printf("ok\n"); fflush(stdout);

    // init data
    printf("Initializing Matrices......."); fflush(stdout);
#endif

	init(n, &hdata, &hmat, &ddata, &dmat1, &dmat2, &msize, &trisize, density);	

#ifdef DEBUG
    printf("ok\n"); fflush(stdout);
#endif

    switch (method){
        case 0:
            r = boundingBox(ddata, dmat1, dmat2, n, nb, rb, msize, trisize);
            break;
        case 1:
            r = lambda(ddata, dmat1, dmat2, n, nb, rb, msize, trisize);
            break;
        case 2:
            r = lambda_tc(ddata, dmat1, dmat2, n, nb, rb, msize, trisize);
            break;
        case 3:
            r = lambda_tc_optimized(ddata, dmat1, dmat2, n, nb, rb, msize, trisize);
            break;
        default:
            printf("Method can only take values 0, 1, 2 or 3\n");
            exit(2);
    }
    //hipMemcpy(Ec_h, Ec_d, sizeof(MTYPE)*n*n, hipMemcpyDeviceToHost);
    //hipMemcpy(res_h, res_d, sizeof(MTYPE), hipMemcpyDeviceToHost);

#ifdef DEBUG
    printf("gpudummy(): verifying results..."); fflush(stdout);
#endif

    //assert(verifyCA(n, msize, hdata, ddata, hmat, dmat2));

#ifdef DEBUG
    printf("ok\n\n"); fflush(stdout);
#endif

    free(hdata);
    free(hmat);
    hipFree(ddata);
    hipFree(dmat1);
    hipFree(dmat2);

    // Return computing time
    stat.mean = r->Mean();
    stat.variance = r->Variance();
    stat.stdev = r->StandardDeviation();
    stat.sterr = r->StandardDeviation()/((double)sqrt(r->NumDataValues()));

    free(r);
    return stat;
}


RunningStat* boundingBox(MTYPE *ddata, MTYPE *dmat1, MTYPE *dmat2, unsigned long n, unsigned long nb, unsigned long rb, unsigned long msize, unsigned long trisize){

    dim3 block, grid;

    auto psgen = [] (unsigned long rb, int BSIZE, dim3 &b, dim3 &g){ 
        b = dim3(BSIZE, BSIZE, 1);
        g = dim3(pow(2,rb), pow(2,rb), 1);
    };

    auto bbmap = [] __device__ (const int nb, const int rb, const int WSIZE, half *mata, half *matb, float *matc){
        uint2 m;
        m.x = blockIdx.x*blockDim.x + threadIdx.x;
        m.y = blockIdx.y*blockDim.y + threadIdx.y;
        return m;
    };
    
    psgen(rb, 1<<BPOWER, block, grid);

    return performLoad(ddata, dmat1, dmat2, n, nb, rb, msize, trisize, block, grid, bbmap, 0, 0, 0);
}

RunningStat* lambda(MTYPE *ddata, MTYPE *dmat1, MTYPE *dmat2, unsigned long n, unsigned long nb, unsigned long rb, unsigned long msize, unsigned long trisize){

    dim3 block, grid;

    // pspace: orthotope for lambda
    auto psgen =  [] (unsigned long rb, int BSIZE, dim3 &b, dim3 &g){ 
        b = dim3(BSIZE, BSIZE, 1);
        g = dim3((int)pow(3, ceil(rb/2.0)), (int)pow(3, floor(rb/2.0)), 1); 
    };

    // lambda map
    auto lambdamap = [] __device__ (const int nb, const int rb, const int WSIZE, half *mata, half *matb, float *matc){
        __shared__ uint2 m;
        auto beta = [] __device__ (const int nb, const uint2 w, const int u){
            int b = (int)((blockIdx.x*(u & 1) + blockIdx.y*((u+1) & 1))/(pow3((u>>1) + (u&1) - 1)))%3;
            return b;
        };
        int lid = threadIdx.x + threadIdx.y*blockDim.x;
        int tid = lid;
        if(lid < WSIZE){
            uint2 lm = {0,0};
            while(lid < rb){
                int b = beta(nb, {blockIdx.x, blockIdx.y}, lid+1);
                lm.x += (b >> 1) * (1 << (lid));
                lm.y += (b - (b >> 1)) * (1 << (lid));
                lid += WSIZE;
            }
            lm = warp_reduce(lm,WSIZE); 
            if(tid == 0){ m = lm; }
        }
        __syncthreads();
        return (uint2){m.x * blockDim.x + threadIdx.x, m.y * blockDim.y + threadIdx.y};
    };

    psgen(rb, 1<<BPOWER, block, grid);

    return performLoad(ddata, dmat1, dmat2, n, nb, rb, msize, trisize, block, grid, lambdamap, 0, 0, 0);
}

RunningStat* lambda_tc(MTYPE *ddata, MTYPE *dmat1, MTYPE *dmat2, unsigned long n, unsigned long nb, unsigned long rb, unsigned long msize, unsigned long trisize){

    dim3 block, grid;

    if (BSIZE1D < 32){
#ifdef DEBUG
        printf("Blocksize needs to be at least 32 to use warp sync mma operations.\n");
#endif
        return nullptr;
    }

    // pspace: orthotope for lambda
    auto psgen = [] (unsigned long rb, int BSIZE, dim3 &b, dim3 &g){ 
        b = dim3(BSIZE, BSIZE, 1);
        g = dim3((int)pow(3, ceil(rb/2.0)), (int)pow(3, floor(rb/2.0)), 1); 
    };

    // Tensor core lambda map
    // This map assumes that the block size is >=32, which is the minimum to perform tensor core mma,
    auto lambdamap_tc = [] __device__ (const int nb, const int rb, const int WSIZE, half *mata, half *matb, float *matc){
        
        //Has to be declared after the matrices above to avoid 8-byte shifting
        __shared__ uint2 m;

        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_fragment;
        wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_fragment;
        wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_fragment;

        auto beta = [] __device__ (const int nb, const int u){
            int b = (int)((blockIdx.x*(u & 1) + blockIdx.y*((u+1) & 1))/(pow3((u>>1) + (u&1) - 1)))%3;
            return b;
        };
        
        int lid = threadIdx.x + threadIdx.y*blockDim.x;
        int index = lid;

        if (lid < 32) {
            //Has to be resetted to 0. Latter kernel calls were getting weird values
            #pragma unroll
            for (int i=0; i<2; i++){
                matb[i*32 + lid] = 0;
            }
            if (lid < rb){
                mata[lid] = 1 << lid;
            }
        } else {
            lid -= 32;
            if (lid < rb*2){
                char column = lid/rb;
                //lid = lid%rb;
                lid = lid-rb*column;
                char b = beta(nb, lid+1);
                int lm = ((b-(b >> 1))*(column & 1) + (b >> 1)*((column+1) & 1));
                matb[lid + column*16] = lm;
            }
        } 

        __syncthreads();
        if (index < 32) {
            wmma::load_matrix_sync(a_fragment, &mata[0], 16);
        
            wmma::load_matrix_sync(b_fragment, &matb[0], 16);
            wmma::mma_sync(c_fragment, a_fragment, b_fragment, c_fragment);
        }

        __syncthreads();
        if (index == 0){
            m = (uint2){(int)(c_fragment.x[0]), (int)(c_fragment.x[1])};
        }

        __syncthreads();

        return (uint2){m.x * blockDim.x + threadIdx.x, m.y * blockDim.y + threadIdx.y};
    };

    psgen(rb, 1<<BPOWER, block, grid);

    return performLoad(ddata, dmat1, dmat2, n, nb, rb, msize, trisize, block, grid, lambdamap_tc, 0, 0, 0 );
}

RunningStat* lambda_tc_optimized(MTYPE *ddata, MTYPE *dmat1, MTYPE *dmat2, unsigned long n, unsigned long nb, unsigned long rb, unsigned long msize, unsigned long trisize){

    dim3 block, grid;

    rb++;


    if (BPOWER != 5){
#ifdef DEBUG
        printf("BPOWER has to be 5 to use this method.\n");
#endif
        return nullptr;
    }

    // pspace: orthotope for lambda on the thing
    auto psgen = [] (unsigned long rb, int BSIZE, dim3 &b, dim3 &g){ 
        b = dim3(BSIZE, BSIZE, 1);
        g = dim3( (int)ceil(pow(3, ceil(rb/2.0)) / 2.0 ), (int)ceil(pow(3, floor(rb/2.0)) / 2.0), 1); 
    };

    // Tensor core lambda map
    // This map assumes that the block size is >=32, which is the minimum to perform tensor core mma,
    auto lambdamap_tc = [] __device__ (const int nb, const int rb, const int WSIZE, half *mata, half *matb, float *matc){
        
        //Has to be declared after the matrices above to avoid 8-byte shifting
        uint2 m;

        //Strange behaviour when fragments are shared, the entire iteration reads the same value
        wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_fragment;
        wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_fragment;
        wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_fragment;

        wmma::fill_fragment(c_fragment, 0.0f);

        auto beta = [] __device__ (const int nb, uint2 bids, const int u){
            int b = (int)((bids.x*(u & 1) + bids.y*((u+1) & 1))/(pow3((u>>1) + (u&1) - 1)))%3;
            return b;
        };
        
        int lid = threadIdx.x + threadIdx.y*blockDim.x;
        int index = lid;

        if (lid < 256) {
            if (lid < rb){
                mata[lid] = 1 << lid;
            }
        } else {
            lid -= 256;
            if (lid < 128){
                char row = lid >> 4;
                char aux = lid >> 5;
                char b = beta(nb, {(blockIdx.x<<1) + (aux & 1), (blockIdx.y<<1) + (aux>>1)}, (lid&15)+1);
                int lm = ((b-(b >> 1))*(row & 1) + (b >> 1)*((row+1) & 1));
                matb[lid] = lm;
            }
        } 

        __syncthreads();
        if (index < 32) {
            wmma::load_matrix_sync(a_fragment, &mata[0], 16);
        
            wmma::load_matrix_sync(b_fragment, &matb[0], 16);
            wmma::mma_sync(c_fragment, a_fragment, b_fragment, c_fragment);

            wmma::store_matrix_sync(&matc[0], c_fragment, 16, wmma::mem_row_major);
        }
        __syncthreads();

        char x = threadIdx.x >> 4;
        char y = threadIdx.y >> 4;
        
        if ( (blockIdx.x == gridDim.x-1 && x==1) || (blockIdx.y == gridDim.y-1 && y==1)){
            return (uint2){0xFFFFFFFF, 0xFFFFFFFF};
        }
        char ss = (x<<1) + (y<<2);
        m = (uint2){(int)(matc[ss]), (int)(matc[ss+1])};

        return (uint2){(m.x << 4) + (threadIdx.x & 15), (m.y << 4) + (threadIdx.y & 15)};
    };

    psgen(rb, 1<<BPOWER, block, grid);

    return performLoad(ddata, dmat1, dmat2, n, nb, rb, msize, trisize, block, grid, lambdamap_tc, 0, 0, 0);
}


template<typename Lambda>
RunningStat* performLoad( MTYPE *ddata, MTYPE *dmat1, MTYPE *dmat2, unsigned long n, unsigned long nb, unsigned long rb, unsigned long msize, unsigned int trisize, dim3 block, dim3 grid,
                            Lambda map, unsigned int aux1, unsigned int aux2, unsigned int aux3) {


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    // runningstat statistics
    RunningStat *r = new RunningStat();
    float time = 0.0;

    // measure running time
    hipEventRecord(start, 0);	
    for(int k=0; k<REPEATS; k++){
        kernel_update_ghosts<<< (n+BSIZE2D-1)/BSIZE2D, BSIZE2D>>>(n, msize, dmat1, dmat1);
        hipDeviceSynchronize();
        kernel_test<<< grid, block >>>(n, msize, nb, rb, ddata, dmat1, dmat2, map, aux1, aux2, aux3);	
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("result ping\n");
            print_dmat(PRINTLIMIT, n, msize, dmat2, "PING dmat1 -> dmat2");
            getchar();
        #endif

        kernel_update_ghosts<<< (n+BSIZE2D-1)/BSIZE2D, BSIZE2D>>>(n, msize, dmat2, dmat2);
        hipDeviceSynchronize();
        kernel_test<<< grid, block >>>(n, msize, nb, rb, ddata, dmat2, dmat1, map, aux1, aux2, aux3);	
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("result pong\n");
            print_dmat(PRINTLIMIT, n, msize, dmat1, "PONG  dmat2 -> dmat1");
            getchar();
        #endif
    }
#ifdef DEBUG
    printf("done\n"); fflush(stdout);
#endif
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // that's our time!
    last_cuda_error("benchmark-check");

    r->Push(time/(1000.0f * INNER_REPEATS));

	hipEventDestroy(start);
	hipEventDestroy(stop);
    last_cuda_error("benchmark-check");
#ifdef DEBUG
    printf("\x1b[1mok\n\x1b[0m"); fflush(stdout);
#endif

    return r;
}
