#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include "custom_funcs.h"

#define SQRT_COEF 0.0001f
#define REPETITIONS 100
#define MEASURES 10
#define BB 2000
#define OFFSET -0.4999f
//#define OFFSET 0.5f

__device__ double cost_function(float4 *a, int i, int j){
    //return sqrtf( powf(a[i].x-a[j].x, 2) + powf(a[i].y-a[j].y, 2) + powf(a[i].z-a[j].z, 2)) + logf(a[i].w*a[j].w);
    //return sqrtf( powf(a[i].x-a[j].x, 2) + powf(a[i].y-a[j].y, 2) );
    //return sqrtf( (a[i].x-a[j].x)*(a[i].x-a[j].x) );
    return sqrtf((a[i].x-a[j].x)*(a[i].x-a[j].x));
}


void export_result(int *b, int n, const char *filename){
    printf("exporting result vector.......");fflush(stdout);
    FILE *fw = fopen(filename, "w");
    if(!fw){
        fprintf(stderr, "error: cannot write to file %s\n", filename);
        exit(1);
    }
    for(int i=0; i<n*n; i++)
        fprintf(fw, "%i\n", b[i]);
    printf("ok\n");
}


// metodo bounding box
__global__ void bb_method(int *a, int N, int* b, int N2){
	if( blockIdx.x > blockIdx.y )
		return;

	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=j){
         cost_function(a, b, i, j);
    }
}
// metodo td con grid lineal
// este método no es muy bueno, la raiz requiere precision double sino falla muy luego
__global__ void td_method(float4 *a, int N, float* b, int N2){
	int c = (blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x + threadIdx.x;
    if(c<N2){
		unsigned int i = sqrtf(0.25 + 2.0*(float)c)-0.5f;
		unsigned int j = c - (i*(i+1) >> 1);  
        b[c] = cost_function(a, i, j);
	}
}






// metodo td con square root FP32
__global__ void td_method_2dx(float4 *a, int N, float* b, int N2){
    unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
    unsigned int bi = sqrtf(0.25 + 2.0f*(float)bc) - 0.5f;
    unsigned int bj = bc - (bi*(bi+1) >> 1);
    
    int i = bi * blockDim.y + threadIdx.y;
	int j = bj * blockDim.x + threadIdx.x;
	int c = ((i * (i + 1)) >> 1) + j;
    if(i>=j && c < N2){
        b[c] = cost_function(a, i, j);
    }
}
// metodo td con inversa square root
__global__ void td_method_2dr(float4 *a, int N, float* b, int N2){
    /*
    __shared__ int bi, bj;
    if( threadIdx.y < 2 ){
        unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
        float arg = __fmaf_rn(2.0f, (float)bc, 0.25f);
        unsigned int bi = __fmaf_rn(arg, rsqrt(arg), OFFSET);// + 0.001f;
        unsigned int bj = bc - (bi*(bi+1) >> 1);
    }
    __syncthreads();
    */
    ///* 
    unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
    float arg = __fmaf_rn(2.0f, (float)bc, 0.25f);
    unsigned int bi = __fmaf_rn(arg, rsqrt(arg), OFFSET);// + 0.001f;
    unsigned int bj = bc - (bi*(bi+1) >> 1);
    //*/
    unsigned int i = bi * blockDim.y + threadIdx.y;
	unsigned int j = bj * blockDim.x + threadIdx.x;
	int c = ((i * (i + 1)) >> 1) + j;
    if(i>=j && c < N2){
         b[c] = cost_function(a, i, j);
    }
}


// metodo rectangle
__global__ void rect_method(float4 *a, int N, float *b, int N2){

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= N+1 || j >= N/2)
        return;

    if( j >= i ){
        j = N - j -1;
        i = N - i -1;
    }
    else
        i = i-1;

    int c = ((i*(i+1)) >> 1) + j;
    if( c < N2 ){
        b[c] = cost_function(a, i, j);
    }   

}


// metodo recursivo
__global__ void recursive_method(float4 *a, int N, float *b, int N2, int bx, int by){

    // calcula el indice del bloque recursivo, es division entera.
    int rec_index = blockIdx.x/gridDim.y;
    
    // calcula el offset, el valor x del bloque respecto al recblock que le corresponde.
    int dbx = blockIdx.x % gridDim.y;
    int tx = (bx+(gridDim.y*rec_index*2) + dbx)*blockDim.x + threadIdx.x;
    int ty = (by+(gridDim.y*rec_index*2) + blockIdx.y)*blockDim.y + threadIdx.y;

    int c = ((ty*(ty+1)) >> 1) + tx;
    if(c < N2){
        b[c] = cost_function(a, ty, tx);
    }
}

__global__ void recursive_diagonal(float4 *a, int N, float *b, int N2){

    // calcula el indice del bloque recursivo, es division entera.
    int rec_index = blockIdx.x/gridDim.y;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = (blockIdx.y + rec_index*gridDim.y)*blockDim.y + threadIdx.y;
    if( tx > ty )
        return;
    int c = ((ty*(ty+1)) >> 1) + tx;
    if( c < N2 ){
         b[c] = cost_function(a, ty, tx);
    }
}


void fill_random(int *array, int n){
	for(int i=0; i<n*n; i++){
		array[i] = (int)(0.5f + (float)rand()/(float)RAND_MAX);
    }
} 

void td_computationn(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_2dn....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method_2dn <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/(REPETITIONS-1);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - (accum*accum)/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("newton_results.dat", "a");
    fprintf(fa, "%i       %f      %f\n", N, mean, stdev/mean);
    fclose(fa);
}


void bb_computation(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){

    //printf("entering bb...\n");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel bb_method....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int m=0; m<MEASURES; m++){
            bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("bb_results.dat", "a");
    fprintf(fa, "%i        %f      %f\n", N, mean, stdev/mean);
    fclose(fa);
}
void td_computationx(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_2dx....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method_2dx <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("exactsqrt_results.dat", "a");
    fprintf(fa, "%i        %f         %f\n", N, mean, stdev/mean);
    fclose(fa);
}

void td_computation_rsqrt(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_rsqrt....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	td_method_2dr <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method_2dr <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("rsqrt_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);

}

void td_computation_rectangle(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel rect_method....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	rect_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            rect_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("rectangle_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);

}

void td_computation_simple(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_1dgrid....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	td_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("grid1d_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);

}

void td_computation_recursive(float *b_h, size_t sizeb, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2, int m, int kval){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_recursive....block= %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z);
    //warmup
    printf("warmup...."); fflush(stdout);
    int bx=0, by=0;
    int bm = m/dimblock.x;
	for(int r=0; r<MEASURES; r++){
        by = 0;
        dim3 dg_diag(N/dimblock.x, bm, 1);
        recursive_diagonal <<< dg_diag, dimblock >>> (a_d, N, b_d, N2);
        by += bm;
        for(int i=0; i<kval; i++){
            dim3 dimgrid(N/(dimblock.x*2), bm*pow(2, i), 1);
            recursive_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2, bx, by);	
            hipDeviceSynchronize();
            by += bm*pow(2,i);
        }
    }
    
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", 
            REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, 
            stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            by = 0;
            dim3 dg_diag(N/dimblock.x, bm, 1);
            recursive_diagonal <<< dg_diag, dimblock >>> (a_d, N, b_d, N2);
            by += bm;
            for(int i=0; i<kval; i++){
                dim3 dimgrid(N/(dimblock.x*2), bm*pow(2, i), 1);
                recursive_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2, bx, by);	
                hipDeviceSynchronize();
                by += bm*pow(2,i);
            }
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else{
		printf("OK!\n");
    }
    
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    
    FILE *fa;
    fa = fopen("recursive_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);
    

}


void print_results(float *b, int n){
	for(int i=0; i<n; i++)
		printf("b[%d] = %f\n", i, b[i]);
}

// main routine that executes on the host
int main(int argc, char **argv){
	//srand ( time(NULL) );
	if( argc < 2 ){
		printf("arguments must be: <N> <method> <extra arg> <outfile>\n");
		exit(1);
	}

    int *a_h, *a_d;
	int *b_h, *b_d;
	const int N = atoi(argv[1]);
	size_t size = N * N * sizeof(float);
	
	printf("doing a td-problem of: %ix%i\n", N, N);
	a_h = (int*)malloc(size);
	b_h = (int*)malloc(size);

	fill_random(a_h, N, N);
	hipMalloc((void **) &a_d, size);
	hipMalloc((void **) &b_d, size);
    hipMemcpy(b_d, b_h1, sizeb, hipMemcpyHostToDevice);
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	
	dim3 dimblock1(16, 16, 1);
	dim3 dimgrid1((N+dimblock.x -1)/dimblock.x, (N+dimblock.x-1)/dimblock.y, 1);	

	int sn = (N+dimblock.x-1)/dimblock.x;
	int sd = sn*(sn+1)/2;
	int s = ceil(sqrt((double)sd));	
   	dim3 dimgrid3(s, s, 1);
    int rect_evenx = N/2;
    int rect_oddx = (int)ceil((float)N/2.0f);
    dim3 dbrect(16, 16, 1);
    dim3 dgrecteven((rect_evenx+dbrect.x-1)/dbrect.x, ((N+1)+dbrect.y-1)/dbrect.y, 1);
    dim3 dgrectodd((rect_oddx+dbrect.x-1)/dbrect.x, (N+dbrect.y-1)/dbrect.y, 1);
	printf("\n");
	//compare_results(b_h1, b_h2, b_h3, N2);
    if(atoi(argv[2])==1){
	    bb_computation(b_h, size, dimgrid1, dimblock, a_d, N, b_d, N);
        if(argc >= 4)
            export_result(b_h, N, argv[3]);
    }
    else if(atoi(argv[2])==7){
        if(N%2==0)
            td_computation_rectangle(b_h, size, dgrecteven, dbrect, a_d, N, b_d, N);
        else
            td_computation_rectangle(b_h, size, dgrectodd, dbrect, a_d, N, b_d, N);

        if(argc >= 4)
            export_result(b_h, N, argv[3]);
    }
    else if(atoi(argv[2])==6){
        td_computation_rsqrt(b_h3, sizeb, dimgrid3, dimblock, a_d, N, b_d, N);
        if(argc >= 4)
            export_result(b_h3, N2, argv[3]);

    }
    else if(atoi(argv[2])==8){
        int n=atoi(argv[3]);
        int m=N/n;
        if( (m % dimblock.x) != 0 ){
            fprintf(stderr, "error: m=%i, not a multiple of %i\n", m, dimblock.x);
            exit(1);
        }
        int k=cf_log2i(n);
        printf("N(dim)=%i  n=%i  m=%i\n", N, n, m);
        td_computation_recursive(b_h, sizeb, dimblock, a_d, N, b_d, N, m, k);
        if(argc >= 5){
            export_result(b_h, N, argv[4]);
        }
    }
    printf("cleaning memory.......");fflush(stdout);
	free(a_h); 
    free(b_h);
	hipFree(a_d);
	hipFree(b_d);
    printf("ok\n");
}

