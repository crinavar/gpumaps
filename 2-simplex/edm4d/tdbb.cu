#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include "custom_funcs.h"

#define SQRT_COEF 0.0001f
#define REPETITIONS 100
#define MEASURES 10
#define BB 2000
#define OFFSET -0.4999f
//#define OFFSET 0.5f

__device__ double cost_function(float4 *data, int i, int j){
    //return sqrtf( powf(a[i].x-a[j].x, 2) + powf(a[i].y-a[j].y, 2) + powf(a[i].z-a[j].z, 2)) + logf(a[i].w*a[j].w);
    //return sqrtf( powf(a[i].x-a[j].x, 2) + powf(a[i].y-a[j].y, 2) );
    //return sqrtf( (a[i].x-a[j].x)*(a[i].x-a[j].x) );
    float4 a = data[i];
    float4 b = data[j];
    return sqrtf( (a.x-b.x)*(a.x-b.x)  + (a.y-b.y)*(a.y-b.y) + (a.z-b.z)*(a.z-b.z) + (a.w-b.w)*(a.w-b.w));
}


void export_result(float *b, int n, const char *filename){
    printf("exporting result vector.......");fflush(stdout);
    FILE *fw = fopen(filename, "w");
    if(!fw){
        fprintf(stderr, "error: cannot write to file %s\n", filename);
        exit(1);
    }
    for(int i=0; i<n; i++)
        fprintf(fw, "%f\n", b[i]);
    printf("ok\n");
}


__device__ float carmack_sqrtf(float nb) {
    float nb_half = nb * 0.5F;
    float y = nb;
    long i = * (long *) &y;
    //i = 0x5f3759df - (i >> 1);
    i = 0x5f375a86 - (i >> 1);
    y = * (float *) &i;
    //Repetitions increase accuracy(6)
    y = y * (1.5f - (nb_half * y * y));
    y = y * (1.5f - (nb_half * y * y));
    y = y * (1.5f - (nb_half * y * y));

    return nb * y;
}
__device__ inline float newton_sqrtf(const float number) {
    int i;
    float x,y;
    //const float f = 1.5F;
    x = number * 0.5f;
    i  = * ( int * ) &number;
    i  = 0x5f3759df - ( i >> 1 );
    y  = * ( float * ) &i;
    y  *= (1.5f -  x * y * y);
    y  *= (1.5f -  x * y * y); 
    y  *= (1.5f -  x * y * y); 
    return number * y;
}

__device__ inline float newton1_sqrtf(const float number){

    int i;
    float x,y;
    //const float f = 1.5F;
    x = number * 0.5f;
    i  = * ( int * ) &number;
    i  = 0x5f3759df - ( i >> 1 );
    y  = * ( float * ) &i;
    y *= (1.5f -  x * y * y);
    y *= number; //obteniendo resultado
    //arreglar
    if( (y+1.0f)*(y+1.0f) < number)
        return y;
    else
        return y-0.5f;
}

// metodo newton
__global__ void td_method_2dn(float4 *a, int N, float* b, int N2){
	unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
	unsigned int bi = newton_sqrtf(0.25f + 2.0f*(float)bc) - 0.5f + SQRT_COEF*(float)1.0f;
	unsigned int bj = (bc - (bi*(bi+1) >> 1));

	int i = bi * blockDim.y + threadIdx.y;
	int j = bj * blockDim.x + threadIdx.x;
    unsigned int c = ((i*(i+1)) >> 1) + j;
    if(i>= j && c<N2){
        b[c] = cost_function(a, i, j);
    }
}

// metodo bounding box
__global__ void bb_method(float4 *a, int N, float* b, int N2){
	if( blockIdx.x > blockIdx.y )
		return;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int c = ((i * (i + 1)) >> 1) + j;
	if(i>=j && c<N2){
	     b[c] = cost_function(a, i, j);
    }
}


// metodo avril map
__global__ void td_method(float4 *data, int N, float* bdata, int N2){
	int k = (blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x + threadIdx.x;
    if(k<N2){
		//int a = __fadd_rn((float)N, __fadd_rn(0.5f, - (carmack_sqrtf(__fmaf_rn((float)N, (float)N, -(float)N) + __fmaf_rn(2.0f, (float)k, 0.25f)))));
	    //int b = (a+1) + k - (((a-1)*(2*N-a)) >> 1);  
        int a = (-(2.0f*(float)N + 1.0f) + carmack_sqrtf(4.0f*(float)N*(float)N - 4.0f*(float)N - 8.0f*(float)k + 1.0f))/(-2.0f);
        int b = (a+1) + k - ((a-1)*(2*N-a))/2;
        if(b > N){
            a++;
            b = a+(b-N);
        }
        if(a >= b){
            b = N - (a-b);
            a--;
        }
        //printf("writing into %i %i\n", a, b);
		// transpose
        bdata[k] = cost_function(data, a, b);
		//bdata[((b*(b+1)) >> 1) + a] = cost_function(data, a, b);
	}
}






// metodo td con square root FP32
__global__ void td_method_2dx(float4 *a, int N, float* b, int N2){
    unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
    unsigned int bi = sqrtf(0.25 + 2.0f*(float)bc) - 0.5f;
    unsigned int bj = bc - (bi*(bi+1) >> 1);
    
    int i = bi * blockDim.y + threadIdx.y;
	int j = bj * blockDim.x + threadIdx.x;
	int c = ((i * (i + 1)) >> 1) + j;
    if(i>=j && c < N2){
        b[c] = cost_function(a, i, j);
    }
}
// metodo td con inversa square root
__global__ void td_method_2dr(float4 *a, int N, float* b, int N2){
    /*
    __shared__ int bi, bj;
    if( threadIdx.y < 2 ){
        unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
        float arg = __fmaf_rn(2.0f, (float)bc, 0.25f);
        unsigned int bi = __fmaf_rn(arg, rsqrt(arg), OFFSET);// + 0.001f;
        unsigned int bj = bc - (bi*(bi+1) >> 1);
    }
    __syncthreads();
    */
    ///* 
    unsigned int bc = blockIdx.x + blockIdx.y*gridDim.x;
    float arg = __fmaf_rn(2.0f, (float)bc, 0.25f);
    unsigned int bi = __fmaf_rn(arg, rsqrtf(arg), OFFSET);// + 0.001f;
    unsigned int bj = bc - (bi*(bi+1) >> 1);
    //*/
    unsigned int i = bi * blockDim.y + threadIdx.y;
	unsigned int j = bj * blockDim.x + threadIdx.x;
	int c = ((i * (i + 1)) >> 1) + j;
    if(i>=j && c < N2){
         b[c] = cost_function(a, i, j);
    }
}


// metodo rectangle
__global__ void rect_method(float4 *a, int N, float *b, int N2){

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= N+1 || j >= N/2)
        return;

    if( j >= i ){
        j = N - j -1;
        i = N - i -1;
    }
    else
        i = i-1;

    int c = ((i*(i+1)) >> 1) + j;
    if( c < N2 ){
        b[c] = cost_function(a, i, j);
    }   

}


// metodo recursivo
__global__ void recursive_method(float4 *a, int N, float *b, int N2, int bx, int by){

    // calcula el indice del bloque recursivo, es division entera.
    int rec_index = blockIdx.x/gridDim.y;
    
    // calcula el offset, el valor x del bloque respecto al recblock que le corresponde.
    int dbx = blockIdx.x % gridDim.y;
    int tx = (bx+(gridDim.y*rec_index*2) + dbx)*blockDim.x + threadIdx.x;
    int ty = (by+(gridDim.y*rec_index*2) + blockIdx.y)*blockDim.y + threadIdx.y;

    int c = ((ty*(ty+1)) >> 1) + tx;
    if(c < N2){
        b[c] = cost_function(a, ty, tx);
    }
}

__global__ void recursive_diagonal(float4 *a, int N, float *b, int N2){

    // calcula el indice del bloque recursivo, es division entera.
    int rec_index = blockIdx.x/gridDim.y;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = (blockIdx.y + rec_index*gridDim.y)*blockDim.y + threadIdx.y;
    if( tx > ty )
        return;
    int c = ((ty*(ty+1)) >> 1) + tx;
    if( c < N2 ){
         b[c] = cost_function(a, ty, tx);
    }
}

int* load_table(const char *filename, int *n){
    int *table;
    FILE *fr = fopen(filename, "rb");
    if(!fr){
        printf("error loading table.... does file '%s' exist?\n", filename);
        exit(-1);
    }
    fread(n, sizeof(int), 1, fr);
    table = (int*)malloc((*n)*sizeof(int));
    fread(table, sizeof(int), *n, fr);
    return table;
}



void fill_random(float4 *array, int n){
	for(int i=0; i<n; i++){
		array[i].x = 100.0f * (float)rand()/(float)RAND_MAX;
        array[i].y = 100.0f * (float)rand()/(float)RAND_MAX;
        array[i].z = 100.0f * (float)rand()/(float)RAND_MAX;
        array[i].w = 100.0f * (float)rand()/(float)RAND_MAX;
    }
} 

void td_computationn(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_2dn....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method_2dn <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/(REPETITIONS-1);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - (accum*accum)/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("newton_results.dat", "a");
    fprintf(fa, "%i       %f      %f\n", N, mean, stdev/mean);
    fclose(fa);
}


void bb_computation(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){

    //printf("entering bb...\n");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel bb_method....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int m=0; m<MEASURES; m++){
            bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("bb_results.dat", "a");
    fprintf(fa, "%i        %f      %f\n", N, mean, stdev/mean);
    fclose(fa);
}
void td_computationx(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_2dx....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	bb_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method_2dx <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("exactsqrt_results.dat", "a");
    fprintf(fa, "%i        %f         %f\n", N, mean, stdev/mean);
    fclose(fa);
}

void td_computation_rsqrt(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_rsqrt....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	td_method_2dr <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method_2dr <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("rsqrt_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);

}

void td_computation_rectangle(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel rect_method....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	rect_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            rect_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("rectangle_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);

}

void td_computation_simple(float *b_h, size_t sizeb, dim3 dimgrid, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel avril method....block= %i x %i x %i    grid = %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z, dimgrid.x, dimgrid.y, dimgrid.z);
    //warmup
    printf("warmup...."); fflush(stdout);
	for(int i=0; i<MEASURES; i++){
    	td_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
        hipDeviceSynchronize();
    }
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            td_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2);	
            hipDeviceSynchronize();
        }

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else
		printf("OK!\n");
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    FILE *fa;
    fa = fopen("avril_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);

}

void td_computation_recursive(float *b_h, size_t sizeb, dim3 dimblock, float4 *a_d, int N, float *b_d, int N2, int m, int kval){
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	printf("calling kernel td_method_recursive....block= %i x %i x %i\n", dimblock.x, dimblock.y, dimblock.z);
    //warmup
    printf("warmup...."); fflush(stdout);
    int bx=0, by=0;
    int bm = m/dimblock.x;
	for(int r=0; r<MEASURES; r++){
        by = 0;
        dim3 dg_diag(N/dimblock.x, bm, 1);
        recursive_diagonal <<< dg_diag, dimblock >>> (a_d, N, b_d, N2);
        by += bm;
        for(int i=0; i<kval; i++){
            dim3 dimgrid(N/(dimblock.x*2), bm*pow(2, i), 1);
            recursive_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2, bx, by);	
            hipDeviceSynchronize();
            by += bm*pow(2,i);
        }
    }
    
    printf("ok\nmeasuring mean time of %i averages (each averages is computed from %i measures).......\n", 
            REPETITIONS, MEASURES); fflush(stdout);
    float accum=0.0f, elapsedTime=0.0f, squared_accum=0.0f, mean=0.0f, time=0.0f, 
            stdev=0.0f, div=(float)((long)REPETITIONS*((long)REPETITIONS-1));
    for(int j=0; j<REPETITIONS; j++){
        hipEventRecord(start, 0);	
        for(int k=0; k<MEASURES; k++){
            by = 0;
            dim3 dg_diag(N/dimblock.x, bm, 1);
            recursive_diagonal <<< dg_diag, dimblock >>> (a_d, N, b_d, N2);
            by += bm;
            for(int i=0; i<kval; i++){
                dim3 dimgrid(N/(dimblock.x*2), bm*pow(2, i), 1);
                recursive_method <<< dimgrid, dimblock >>> (a_d, N, b_d, N2, bx, by);	
                hipDeviceSynchronize();
                by += bm*pow(2,i);
            }
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
        time = elapsedTime/(float)MEASURES;
        accum += time;
        squared_accum += time*time/((float)REPETITIONS-1.0);
        if( j%100 == 0){ printf("."); fflush(stdout);}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
    mean = accum/(float)REPETITIONS;
    stdev = sqrtf(squared_accum - accum*accum/div);
	printf("\nAverage of %i averages(%i):\ncudaEventElapsedTime:\tmean=%f[ms]    stdev=%f    error=%f%%\n", 
    REPETITIONS, MEASURES, mean, stdev, 100.0f*stdev/mean);
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	else{
		printf("OK!\n");
    }
    
	hipMemcpy(b_h, b_d, sizeb, hipMemcpyDeviceToHost);
    
    FILE *fa;
    fa = fopen("recursive_results.dat", "a");
    fprintf(fa, "%i        %f       %f\n", N, mean, stdev/mean);
    fclose(fa);
    

}


void print_results(float *b, int n){
	for(int i=0; i<n; i++)
		printf("b[%d] = %f\n", i, b[i]);
}

// main routine that executes on the host
int main(int argc, char **argv){
	//srand ( time(NULL) );
	if( argc < 2 ){
		printf("arguments must be: <N> <method: 0 (bb), 1 (td)>\n");
		exit(1);
	}

    float4 *a_h, *a_d;
	float *b_h, *b_d;
	const int N = atoi(argv[1]);
	const int N2 = N*(N+1)/2;
    const int Na = N*(N-1)/2;
	size_t size = N * sizeof(float4);
	size_t sizeb = sizeof(float)*N2;
	
	
	printf("doing a td-problem of:\nN=%i\nN2=%i\n", N, N2);
	a_h = (float4*)malloc(size);
	b_h = (float*)malloc(sizeb);
    for(int i=0; i<N2; i++)
        b_h[i] = -1.0f;


	fill_random(a_h, N);
	hipMalloc((void **) &a_d, size);
	hipMalloc((void **) &b_d, sizeb);
    hipMemcpy(b_d, b_h, sizeb, hipMemcpyHostToDevice);
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

	
	dim3 dimblock1(16, 16, 1);
	dim3 dimgrid1(N/dimblock1.x + (N%dimblock1.x == 0 ? 0:1), N/dimblock1.y + (N%dimblock1.y == 0 ? 0:1), 1);	

	dim3 dimblock2(256, 1, 1);
	int sn2 = (Na+dimblock2.x-1)/dimblock2.x;
	int s2 = ceil(sqrt((double)sn2));
    dim3 dimgrid2(s2, s2, 1);
	dim3 dimblock3(16, 16, 1);
	int sn = (N+dimblock3.x-1)/dimblock3.x;
	int sd = sn*(sn+1)/2;
	int s = ceil(sqrt((double)sd));	
	//dim3 dimgrid3((int)sd, 1, 1);	
   	dim3 dimgrid3(s, s, 1);
    int rect_evenx = N/2;
    int rect_oddx = (int)ceil((float)N/2.0f);
    dim3 dbrect(16, 16, 1);
    dim3 dgrecteven((rect_evenx+dbrect.x-1)/dbrect.x, ((N+1)+dbrect.y-1)/dbrect.y, 1);
    dim3 dgrectodd((rect_oddx+dbrect.x-1)/dbrect.x, (N+dbrect.y-1)/dbrect.y, 1);
	printf("\n");
	//compare_results(b_h1, b_h2, b_h3, N2);
    if(atoi(argv[2])==1){
	    bb_computation(b_h, sizeb, dimgrid1, dimblock1, a_d, N, b_d, N2);
        if(argc >= 4)
            export_result(b_h, N2, argv[3]);
    }
    else if(atoi(argv[2])==2){
	    td_computation_simple(b_h, sizeb, dimgrid2, dimblock2, a_d, N, b_d, Na);
        if(argc >= 4)
            export_result(b_h, Na, argv[3]);
    }
    else if(atoi(argv[2])==3){
	    td_computationn(b_h, sizeb, dimgrid3, dimblock3, a_d, N, b_d, N2);
        if(argc >= 4)
            export_result(b_h, N2, argv[3]);
    }
    else if(atoi(argv[2])==4){
        td_computationx(b_h, sizeb, dimgrid3, dimblock3, a_d, N, b_d, N2);
        if(argc >= 4)
            export_result(b_h, N2, argv[3]);
    }
    else if(atoi(argv[2])==7){
        if(N%2==0)
            td_computation_rectangle(b_h, sizeb, dgrecteven, dbrect, a_d, N, b_d, N2);
        else
            td_computation_rectangle(b_h, sizeb, dgrectodd, dbrect, a_d, N, b_d, N2);

        if(argc >= 4)
            export_result(b_h, N2, argv[3]);
    }
    else if(atoi(argv[2])==6){
        td_computation_rsqrt(b_h, sizeb, dimgrid3, dimblock3, a_d, N, b_d, N2);
        if(argc >= 4)
            export_result(b_h, N2, argv[3]);

    }
    else if(atoi(argv[2])==8){
        int n=atoi(argv[3]);
        int m=N/n;
        if( (m % dimblock3.x) != 0 ){
            fprintf(stderr, "error: m=%i, not a multiple of %i\n", m, dimblock3.x);
            exit(1);
        }
        int k=cf_log2i(n);
        printf("N(dim)=%i  n=%i  m=%i\n", N, n, m);
        td_computation_recursive(b_h, sizeb, dimblock3, a_d, N, b_d, N2, m, k);
        if(argc >= 5){
            export_result(b_h, N2, argv[4]);
        }
    }
    printf("cleaning memory.......");fflush(stdout);
	free(a_h); 
    free(b_h);
	hipFree(a_d);
	hipFree(b_d);
    printf("ok\n");
}

