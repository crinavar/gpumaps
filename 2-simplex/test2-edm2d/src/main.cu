#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include <stdint.h>
#include "custom_funcs.h"
#include "kernels.cuh"
#include "gputools.cuh"
#include "gpubenchmarks.cuh"

// main routine that executes on the host
int main(int argc, char **argv){
	//srand ( time(NULL) );
	if(argc < 5){
		printf("arguments must be: <dev> <N> <repeats> <method>\nmethod:\n1 bb\n2 utm\n3 ltmn\n4 ltmx\n5 flatrec\n6 ltmr\n7 rectangle\n8 recursive\n\n");
		exit(1);
	}
    unsigned int dev = atoi(argv[1]);
    unsigned int n = atoi(argv[2]);
    unsigned int REPEATS = atoi(argv[3]);
    unsigned int method = atoi(argv[4]);
    hipSetDevice(dev);
    last_cuda_error("hipSetDevice");
    double time;
    switch(method){
        case 1:
	        time = bbox(n, REPEATS);
            break;
        case 2:
            time = avril(n, REPEATS);
            break;
        case 3:
            time = lambda_newton(n, REPEATS);
            break;
        case 4:
            time = lambda_standard(n, REPEATS);
            break;
        case 5:
            time = lambda_flatrec(n, REPEATS);
            break;
        case 6:
            time = lambda_inverse(n, REPEATS);
            break;
        case 7:
            time = rectangle_map(n, REPEATS);
            break;
        case 8:
            if(argc != 6){
                fprintf(stderr, "recursive map requires an aditional parameter <recn>\n");
                exit(EXIT_FAILURE);
            }
            time = recursive_map(n, atoi(argv[5]), REPEATS);
            break;
    }
    printf("%f\n", time);
}

